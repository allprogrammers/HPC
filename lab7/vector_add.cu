
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to add two vectors

__global__ void vectorAdd(int* a, int* b, int* c, int n)
{
    // Get the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if the thread ID is within the vector size
    if (tid<n)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main(void)
{
    int n = 10000; //Vector size
    int* a; // Host cectors
    int* b;
    int* c;
    int* d_a; //Device vectors
    int* d_b;
    int* d_c;

    // Allocate memory for host vectors
    a = (int*) malloc(n * sizeof(int));
    b = (int*) malloc(n * sizeof(int));
    c = (int*) malloc(n * sizeof(int));
    
    // Initialize host vectors
    for (int i = 0; i < n; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    // Allocate memeory for device vectors
    hipMalloc(&d_a,n*sizeof(int));
    hipMalloc(&d_b,n*sizeof(int));
    hipMalloc(&d_c,n*sizeof(int));
    
    // copy host vectors to device

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions

    int blockSize = 256;
    int gridSize = (n+blockSize -1)/blockSize;

    // Launch the vectorAdd CUDA kernel

    vectorAdd<<<gridSize, blockSize>>>(d_a,d_b,d_c,n);

    // Copy result from device to host

    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    
    // Verify the result

    for(int i=0;i<n;i++)
    {
        if(c[i] != a[i]+b[i])
        {
            printf("Error: Result verification failed at element %d!\n",i);
            return 1;
        }
    }

    printf("Vector addition completed successfully!\n");

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    
    free(a);
    free(b);
    free(c);
    
    return 0;
}