
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>
// CUDA kernel to add two vectors

#define MAX_ELEMENTS 100000

__global__ void vectorAdd(int* a, int* b, int* c, int n)
{
    // Get the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if the thread ID is within the vector size
    if (tid<n)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int readFile(char* filename,int num_elements,int vector[])
{

    //each part of the file can be read by different thread (is it possible?)
    FILE* file = fopen(filename, "r");

    if(file==NULL)
    {
        printf("Error Opening the file.\n");
        return 1;
    }

    for(int i = 0; i<num_elements;i++)
    {
        if (fscanf(file, "%d", &vector[i]) != 1)
        {
            printf("Error reading from the file. \n");
            fclose(file);
            return 1;
        }
    }

    fclose(file);
    return 0;
}

void writeFile(int num_elements,int* vector1, int* vector2, int* result)
{
    FILE* file_avg = fopen("vector_z.txt","w");
    
    for(int i = 0; i<num_elements;i++)
    {
        fprintf(file_avg, "%d %d %d \n", vector1[i], vector2[i],result[i]);
    }
    
    fclose(file_avg);
}

int main(void)
{
    char* env_elements_str = getenv("NUM_ELEMENTS");
    int n;

    if(env_elements_str == NULL)
    {

        printf("Environment variable NUM_ELEMENTS is noto set.\n");

        return 1;

    }

    sscanf(env_elements_str, "%d", &n);

    if(n <= 0 || n > MAX_ELEMENTS)
    {
        printf("Invalid number of elements from environmental variable. \n");
        return 1;
    }

    

    int* a; // Host vectors
    int* b;
    int* c;
    int* d_a; //Device vectors
    int* d_b;
    int* d_c;

    // Allocate memory for host vectors
    a = (int*) malloc(n * sizeof(int));
    b = (int*) malloc(n * sizeof(int));
    c = (int*) malloc(n * sizeof(int));

    readFile("vector_x.txt",n,a);
    readFile("vector_y.txt",n,b);

    
    // Initialize host vectors
    /*for (int i = 0; i < n; i++)
    {
        a[i] = i;
        b[i] = i;
    }*/
    n=n/10;

    omp_set_schedule(omp_sched_auto, 1);
    #pragma omp parallel for
    for(int i = 0;i<10;i++)
    {
        printf("%d\n", omp_get_num_threads());
        hipEvent_t start, stop;
        float elapsedTime;
        hipEventCreate(&start); // create event objects
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        // Allocate memeory for device vectors
        hipMalloc(&d_a,n*sizeof(int));
        hipMalloc(&d_b,n*sizeof(int));
        hipMalloc(&d_c,n*sizeof(int));
        
        // copy host vectors to device

        hipMemcpy(d_a, (a+i*n), n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_b, (b+i*n), n * sizeof(int), hipMemcpyHostToDevice);

        // Define grid and block dimensions

        int blockSize = 256;
        int gridSize = (n+blockSize -1)/blockSize;

        

        // Launch the vectorAdd CUDA kernel

        vectorAdd<<<gridSize, blockSize>>>(d_a,d_b,d_c,n);

        // Copy result from device to host

        hipMemcpy((c+i*n), d_c, n * sizeof(int), hipMemcpyDeviceToHost);

        hipEventRecord(stop, 0); // record end event
        hipEventSynchronize(stop); // wait for all device work to complete
        hipEventElapsedTime(&elapsedTime, start, stop); //time between events
        hipEventDestroy(start); //destroy start event
        hipEventDestroy(stop);

        printf("time elapsed: %f\nthread %d\n", elapsedTime,i);

    }
    
    // Verify the result

    n=n*10;
    for(int i=0;i<n;i++)
    {
        if(c[i] != a[i]+b[i])
        {
            printf("Error: Result verification failed at element %d!\n",i);
            return 1;
        }
    }

    writeFile(n,a,b,c);

    printf("Vector addition completed successfully!\n");

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    
    free(a);
    free(b);
    free(c);
    
    return 0;
}